#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <iostream>
#include <cassert>
#include <numeric>
#include <hip/hip_runtime.h>
#include "../scripts/include/cuda_tools.h"
#include "../scripts/include/nr3.h"
#include "../scripts/include/network.h"
#include "../scripts/include/forwardmethod_jr.h"
#include "../scripts/include/parallel_tempering_kernels.cuh"

void parallel_tempering_cuda_multi(
    int n_replicas, double T_min, double T_max,
    VecDoub_IO &bm_av_s, VecDoub_IO &bm_av_ss,
    int t_eq, int t_step, int relx, int rept,
    int n_spins, double mean, double sigma,
    const int &type, double H,
    std::vector<double> &energy_per_replica,
    std::vector<double> &temperatures,
    double &swap_acceptance_ratio,
    std::mt19937 &gen
) {
    temperatures.resize(n_replicas);
    int idx_T1 = n_replicas / 2;
    temperatures[idx_T1] = 1.0;
    for (int i = 0; i < idx_T1; ++i)
        temperatures[i] = 1.0 * pow(T_min / 1.0, (double)(idx_T1 - i) / idx_T1);
    for (int i = idx_T1 + 1; i < n_replicas; ++i)
        temperatures[i] = 1.0 * pow(T_max / 1.0, (double)(i - idx_T1) / (n_replicas - 1 - idx_T1));

    std::vector<double> betas(n_replicas);
    std::vector<Rede> replicas;
    replicas.reserve(n_replicas);
    for (int i = 0; i < n_replicas; ++i) {
        betas[i] = 1.0 / temperatures[i];
        replicas.emplace_back(n_spins, mean, sigma, betas[i], type, H);
        replicas.back().create_bonds_random();
    }

    int n_bonds = replicas[0].get_all_bonds().size() / 2;
    size_t sz_spins = n_replicas * n_spins * sizeof(int);
    size_t sz_h = n_replicas * n_spins * sizeof(double);
    size_t sz_J = n_replicas * n_bonds * sizeof(double);
    size_t sz_bonds = n_replicas * n_bonds * sizeof(int2);

    int *d_spins_all;
    double *d_h_all, *d_J_all, *d_energies;
    int2 *d_bonds_all;
    double *d_av_s, *d_av_ss;
    hiprandState *d_rand_states;

    hipMalloc(&d_spins_all, sz_spins);
    hipMalloc(&d_h_all, sz_h);
    hipMalloc(&d_J_all, sz_J);
    hipMalloc(&d_bonds_all, sz_bonds);
    hipMalloc(&d_energies, sizeof(double) * n_replicas);
    hipMalloc(&d_av_s, sz_spins);
    hipMalloc(&d_av_ss, sz_J);
    hipMalloc(&d_rand_states, n_replicas * n_spins * sizeof(hiprandState));

    std::vector<int> spins_all(n_replicas * n_spins);
    std::vector<double> h_all(n_replicas * n_spins);
    std::vector<double> J_all(n_replicas * n_bonds);
    std::vector<int2> bonds_all(n_replicas * n_bonds);

    for (int r = 0; r < n_replicas; ++r) {
        const std::vector<int>& bond_indices = replicas[r].get_all_bonds();
        for (int i = 0; i < n_spins; ++i) {
            spins_all[r * n_spins + i] = replicas[r].s[i];
            h_all[r * n_spins + i] = replicas[r].h[i];
        }
        for (int i = 0; i < n_bonds; ++i) {
            J_all[r * n_bonds + i] = replicas[r].J[i];
            int site1 = bond_indices[2 * i];
            int site2 = bond_indices[2 * i + 1];
            bonds_all[r * n_bonds + i] = make_int2(site1, site2);
        }
    }

    hipMemcpy(d_spins_all, spins_all.data(), sz_spins, hipMemcpyHostToDevice);
    hipMemcpy(d_h_all, h_all.data(), sz_h, hipMemcpyHostToDevice);
    hipMemcpy(d_J_all, J_all.data(), sz_J, hipMemcpyHostToDevice);
    hipMemcpy(d_bonds_all, bonds_all.data(), sz_bonds, hipMemcpyHostToDevice);
    hipMemset(d_av_s, 0, sz_spins);
    hipMemset(d_av_ss, 0, sz_J);

    setup_curand<<<n_replicas, n_spins>>>(d_rand_states, gen());

    std::vector<int> indices(n_replicas);
    std::iota(indices.begin(), indices.end(), 0);

    int swap_attempts = 0, swap_accepted = 0;

    for (int rep = 0; rep < rept; ++rep) {
        metropolis_step<<<n_replicas, n_spins>>>(
            d_spins_all, d_h_all, d_J_all, d_bonds_all,
            n_spins, n_bonds, 1.0, d_rand_states, n_replicas
        );
        hipDeviceSynchronize();

        compute_energies_kernel<<<n_replicas, 1>>>(
            d_spins_all, d_h_all, d_J_all, d_bonds_all,
            d_energies, n_spins, n_bonds, n_replicas
        );
        hipDeviceSynchronize();

        std::vector<double> energies(n_replicas);
        hipMemcpy(energies.data(), d_energies, sizeof(double) * n_replicas, hipMemcpyDeviceToHost);

        for (int i = 0; i < n_replicas - 1; ++i) {
            int idx_i = indices[i];
            int idx_j = indices[i + 1];
            double delta = (betas[idx_j] - betas[idx_i]) * (energies[idx_j] - energies[idx_i]);
            ++swap_attempts;
            if ((double)gen() / gen.max() < exp(delta)) {
                std::swap(indices[i], indices[i + 1]);
                ++swap_accepted;
            }
        }

        accumulate_observables<<<n_replicas, max(n_spins, n_bonds)>>>(
            d_spins_all, d_av_s, d_av_ss, d_bonds_all,
            n_spins, n_bonds, n_replicas
        );
        hipDeviceSynchronize();
    }

    std::vector<double> av_s_host(n_replicas * n_spins);
    std::vector<double> av_ss_host(n_replicas * n_bonds);

    hipMemcpy(av_s_host.data(), d_av_s, sz_spins, hipMemcpyDeviceToHost);
    hipMemcpy(av_ss_host.data(), d_av_ss, sz_J, hipMemcpyDeviceToHost);

    int target_index = idx_T1;
    for (int i = 0; i < n_spins; ++i)
        bm_av_s[i] = av_s_host[target_index * n_spins + i] / (rept * t_step / relx);

    for (int i = 0; i < n_bonds; ++i)
        bm_av_ss[i] = av_ss_host[target_index * n_bonds + i] / (rept * t_step / relx);

    swap_acceptance_ratio = (swap_attempts > 0) ? (double)swap_accepted / swap_attempts : 0.0;

    hipFree(d_spins_all);
    hipFree(d_h_all);
    hipFree(d_J_all);
    hipFree(d_bonds_all);
    hipFree(d_energies);
    hipFree(d_av_s);
    hipFree(d_av_ss);
    hipFree(d_rand_states);
}





