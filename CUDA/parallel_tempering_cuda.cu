#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "../scripts/include/parallel_tempering_kernels.cuh" // inclui as declarações

__global__ void setup_curand(hiprandState *states, unsigned long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &states[id]);
}

__global__ void metropolis_step(
    int* spins_all, double* h_all, double* J_all, int2* bonds_all,
    int n_spins, int n_bonds, double beta,
    hiprandState *rand_states, int n_replicas
) {
    int rep_id = blockIdx.x;
    if (rep_id >= n_replicas) return;

    int idx = threadIdx.x;
    if (idx >= n_spins) return;

    int spin_offset = rep_id * n_spins;
    int bond_offset = rep_id * n_bonds;

    int s_i = spins_all[spin_offset + idx];
    double dE = 2.0 * s_i * h_all[spin_offset + idx];

    for (int b = 0; b < n_bonds; ++b) {
        int2 bond = bonds_all[bond_offset + b];
        if (bond.x == idx || bond.y == idx) {
            int neighbor = (bond.x == idx) ? bond.y : bond.x;
            dE += 2.0 * J_all[bond_offset + b] * s_i * spins_all[spin_offset + neighbor];
        }
    }

    double r = hiprand_uniform(&rand_states[rep_id * n_spins + idx]);
    if (dE <= 0 || r < exp(-beta * dE)) {
        spins_all[spin_offset + idx] = -s_i;
    }
}

__global__ void accumulate_observables(
    int* spins_all, double* av_s_all, double* av_ss_all, int2* bonds_all,
    int n_spins, int n_bonds, int n_replicas
) {
    int rep_id = blockIdx.x;
    if (rep_id >= n_replicas) return;

    int idx = threadIdx.x;
    int spin_offset = rep_id * n_spins;
    int bond_offset = rep_id * n_bonds;

    if (idx < n_spins)
        atomicAdd(&av_s_all[spin_offset + idx], (double)spins_all[spin_offset + idx]);

    if (idx < n_bonds) {
        int2 bond = bonds_all[bond_offset + idx];
        int si = spins_all[spin_offset + bond.x];
        int sj = spins_all[spin_offset + bond.y];
        atomicAdd(&av_ss_all[bond_offset + idx], (double)(si * sj));
    }
}

__global__ void compute_energies_kernel(
    const int* spins_all, const double* h_all, const double* J_all, const int2* bonds_all,
    double* energies, int n_spins, int n_bonds, int n_replicas
) {
    int idx = blockIdx.x;
    if (idx >= n_replicas) return;

    int spin_offset = idx * n_spins;
    int bond_offset = idx * n_bonds;

    double E = 0.0;

    for (int i = 0; i < n_spins; ++i)
        E -= h_all[spin_offset + i] * spins_all[spin_offset + i];

    for (int i = 0; i < n_bonds; ++i) {
        int2 b = bonds_all[bond_offset + i];
        int s1 = spins_all[spin_offset + b.x];
        int s2 = spins_all[spin_offset + b.y];
        E -= J_all[bond_offset + i] * s1 * s2;
    }

    energies[idx] = E;
}




